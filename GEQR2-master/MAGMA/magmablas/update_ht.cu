#include "hip/hip_runtime.h"
#include "magma_internal.h"
#include "magma_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

//==============================================================================
//==============================================================================
__global__ 
void stemp(float *matrix, float *dot, float *temp)
{
    float beta = sqrt(*dot);
    *temp = -copysign(beta, *matrix);
}

__global__ 
void scnst(int N, float *matrix, int ldda, float *dot, float *temp)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;	

    __syncthreads();

    if(i<N)
    {
	dot[i] = MAGMA_S_DIV(dot[i], temp[0]*(matrix[0]-temp[0])) - MAGMA_S_DIV(matrix[ldda*i], (matrix[0]-temp[0]));
    }

} 

__global__ 
void sdiff(float *matrix, float *diff, float *temp)
{
    *diff = (*matrix - *temp);
}

__global__
void srow1(int N, float *matrix, int ldda, float *dot, float *diff) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;		
 
    float ltemp;
 
    __syncthreads();	

    if(i<N)		
    {
	ltemp = matrix[ldda*i] + MAGMA_S_MUL(dot[i], *diff);
	__syncthreads();
	matrix[ldda*i] = ltemp;
    }

}

__global__
void stmup(int M, float *matrix, int ldda, float *dot, float *vector) 
{
    const int tx = threadIdx.x;

    dot    = dot + blockIdx.x;
    matrix = matrix + blockIdx.x * ldda;

    float tmp;

    if(blockIdx.x != 0){
    	tmp = dot[0];
    	for( int j = M-tx-1; j > 0; j -= BLOCK_SIZE )
	    matrix[j] += tmp*vector[j];
    }
}

__global__
void htcns(int M, float *vector, float *dtau, float *diff) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;		
  
    __syncthreads();	

    if(i == 0)
	*dtau = -(*diff)/(*vector);

    __syncthreads();

    if(i>0 && i<M)		
    {
	vector[i] = vector[i]/(*diff);
    }

}

extern "C" magma_int_t
update_HT(
    hipblasHandle_t handle,
    magma_int_t m,  magma_int_t n,
    magmaFloat_ptr dv,
    magmaFloat_ptr dtau,
    magmaFloat_ptr dC,  magma_int_t lddc,
    magmaFloat_ptr dwork,
    magma_queue_t queue )
{
    #define dC(i_,j_) (dC + (i_) + (j_)*(lddc))

    float alpha, beta;

    alpha  = MAGMA_S_ONE;
    beta   = MAGMA_S_ZERO;

    if( (m % BLOCK_SIZE) != 0 ){
	    dim3 grid    ((m/BLOCK_SIZE)+1,1,1);
    	dim3 threads (BLOCK_SIZE,1,1);

     	hipblasSgemv( handle, cublas_trans_const(MagmaTrans), m, n, &alpha, dC, lddc, dv, 1, &beta, dtau, 1);

 	    stemp<<<  1 ,      1 , 0, queue->hip_stream()>>>(dC(0,0), dtau, dwork);  
      scnst<<<grid, threads, 0, queue->hip_stream()>>>(n, dC(0,0), lddc, dtau, dwork);    
  	  sdiff<<<  1 ,      1 , 0, queue->hip_stream()>>>(dC(0,0), dwork, dwork);  
   	  srow1<<<grid, threads, 0, queue->hip_stream()>>>(n, dC(0,0), lddc, dtau, dwork);    
    	stmup<<<  n , threads, 0, queue->hip_stream()>>>(m, dC(0,0), lddc, dtau, dv);    
    	htcns<<<grid, threads, 0, queue->hip_stream()>>>(m, dv, dtau, dwork);    
   }

    else{
	    dim3 grid    ((m/BLOCK_SIZE),1,1);
    	dim3 threads (BLOCK_SIZE,1,1);
     	
	    hipblasSgemv( handle, cublas_trans_const(MagmaTrans), m, n, &alpha, dC, lddc, dv, 1, &beta, dtau, 1);
 
	    stemp<<<  1 ,      1 , 0, queue->hip_stream()>>>(dC(0,0), dtau, dwork);  
    	scnst<<<grid, threads, 0, queue->hip_stream()>>>(n, dC(0,0), lddc, dtau, dwork);    
   	  sdiff<<<  1 ,      1 , 0, queue->hip_stream()>>>(dC(0,0), dwork, dwork);  
    	srow1<<<grid, threads, 0, queue->hip_stream()>>>(n, dC(0,0), lddc, dtau, dwork);    
    	stmup<<<  n , threads, 0, queue->hip_stream()>>>(m, dC(0,0), lddc, dtau, dv);    
    	htcns<<<grid, threads, 0, queue->hip_stream()>>>(m, dv, dtau, dwork);    
   }

    return MAGMA_SUCCESS;
}

//==============================================================================
